#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

#include <iostream>
#include <vector>

#include "common.h"
#include "kseq/kseq.h"

// Comments written for easier understanding and debugging purposes and revision
// purposes, the main idea is to parallelise the matching of the sample and
// signature sequences using cuda as they are independent tasks

// The cuda kernel to find the match between the sample and the signature
__global__ void matchKernel(char* device_sampleSeq, char* device_signatureSeq,
                            char* device_sampleQual, int* device_samplesSize,
                            int* device_signaturesSize,
                            double* device_matchValue, int sampleNum,
                            int signatureNum, int* device_sampleIdx,
                            int* device_signatureIdx) {
    // Initialising the thread index
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int maxThreads = sampleNum * signatureNum;
    // Comparing bounds
    if (maxThreads <= idx) {
        return;
    } else {
        // Initialising the sample and signature index
        int sampleIdx = idx / signatureNum;
        int signatureIdx = idx % signatureNum;

        // Initialising the match value
        double matchValue = 0;
        int firstCharToMatch = -1;

        // Initialising the sample and signature size
        int sampleSize = device_samplesSize[sampleIdx];
        int signatureSize = device_signaturesSize[signatureIdx];

        // Initialising the sample and signature sequence
        int sampleStartingPosition = device_sampleIdx[sampleIdx];
        char* sampleSeq = &device_sampleSeq[sampleStartingPosition];

        int signatureStartingPosition = device_signatureIdx[signatureIdx];
        char* signatureSeq = &device_signatureSeq[signatureStartingPosition];

        // Initialising the sample quality
        char* sampleQual = &device_sampleQual[sampleStartingPosition];

        // Finding the match(sliding technique)
        // Iterating through the sample
        for (int i = 0; i <= sampleSize - signatureSize; i++) {
            bool isMatch = true;
            // Iterating through the signature
            for (int j = 0; j < signatureSize; j++) {
                // If the character does not match or no wildcards, break the
                // loop
                char sampleChar = sampleSeq[i + j];
                char signatureChar = signatureSeq[j];
                if (sampleChar != signatureChar && sampleChar != 'N' &&
                    signatureChar != 'N') {
                    isMatch = false;
                    break;
                }
            }
            // If match found, store the first character that matches, which is
            // the current index in the sample string
            if (isMatch) {
                firstCharToMatch = i;
                break;
            }
        }
        // If match found, calculate the match value, -1 means didnt match so
        // skip
        if (firstCharToMatch != -1) {
            for (int i = 0; i < signatureSize; i++) {
                char currentQualityAsciiChar = sampleQual[firstCharToMatch + i];
                matchValue += static_cast<double>(currentQualityAsciiChar) - 33;
            }
            matchValue = matchValue / signatureSize;
        }
        // Storing the match value
        device_matchValue[sampleIdx * signatureNum + signatureIdx] = matchValue;
    }
}

// The function to run the cuda matcher
void runMatcher(const std::vector<klibpp::KSeq>& samples,
                const std::vector<klibpp::KSeq>& signatures,
                std::vector<MatchResult>& matches) {
    // 1. Flatten the array of structs in sample and signatures by
    // allocation host memory and copying the data from the structs to the
    // host memory

    // Initialising length of array and maximum size of sample and signature
    int sampleNum = samples.size();
    int signatureNum = signatures.size();
    int numberOfPairs = sampleNum * signatureNum;
    int totalSampleSize = 0;
    int totalSignatureSize = 0;

    // Finding the max size of the sample and signature
    for (int i = 0; i < sampleNum; i++) {
        totalSampleSize += samples[i].seq.size();
    }
    for (int i = 0; i < signatureNum; i++) {
        totalSignatureSize += signatures[i].seq.size();
    }

    // Allocate host memory
    char* host_sampleSeq = (char*)malloc(totalSampleSize * sizeof(char));
    char* host_signatureSeq = (char*)malloc(totalSignatureSize * sizeof(char));
    char* host_sampleQual = (char*)malloc(totalSampleSize * sizeof(char));
    int* host_sampleSize = (int*)malloc(sampleNum * sizeof(int));
    int* host_signatureSize = (int*)malloc(signatureNum * sizeof(int));
    int* host_sampleIdx = (int*)malloc(sampleNum * sizeof(int));
    int* host_signatureIdx = (int*)malloc(signatureNum * sizeof(int));
    double* host_matchValue =
        (double*)malloc(sampleNum * signatureNum * sizeof(double));

    // Copy the data from the structs to the host memory
    int sampleStartIndexes = 0;
    for (int i = 0; i < sampleNum; i++) {
        const char* sampleSeq = samples[i].seq.c_str();
        const char* sampleQual = samples[i].qual.c_str();
        int sampleSize = samples[i].seq.size();
        host_sampleSize[i] = sampleSize;
        host_sampleIdx[i] = sampleStartIndexes;
        for (int j = 0; j < sampleSize; j++) {
            int currentSampIndex = sampleStartIndexes + j;
            host_sampleSeq[currentSampIndex] = sampleSeq[j];
            host_sampleQual[currentSampIndex] = sampleQual[j];
        }
        sampleStartIndexes += sampleSize;
    }

    int signatureStartIndexes = 0;
    for (int i = 0; i < signatureNum; i++) {
        const char* signatureSeq = signatures[i].seq.c_str();
        int signatureSize = signatures[i].seq.size();
        host_signatureSize[i] = signatureSize;
        host_signatureIdx[i] = signatureStartIndexes;
        for (int j = 0; j < signatureSize; j++) {
            int currentSigIndex = signatureStartIndexes + j;
            host_signatureSeq[currentSigIndex] = signatureSeq[j];
        }
        signatureStartIndexes += signatureSize;
    }

    // 2. Then allocate device memory and copy the data from the host memory
    // to the device memory

    // Initialising device memory variable
    char* device_sampleSeq;
    char* device_signatureSeq;
    char* device_sampleQual;
    int* device_samplesSize;
    int* device_signaturesSize;
    int* device_sampleIdx;
    int* device_signatureIdx;
    double* device_matchValue;

    // Allocate device memory using hipMalloc
    hipMalloc(&device_sampleSeq, totalSampleSize * sizeof(char));
    hipMalloc(&device_signatureSeq, totalSignatureSize * sizeof(char));
    hipMalloc(&device_sampleQual, totalSampleSize * sizeof(char));
    hipMalloc(&device_samplesSize, sampleNum * sizeof(int));
    hipMalloc(&device_signaturesSize, signatureNum * sizeof(int));
    hipMalloc(&device_sampleIdx, sampleNum * sizeof(int));
    hipMalloc(&device_signatureIdx, signatureNum * sizeof(int));
    hipMalloc(&device_matchValue, numberOfPairs * sizeof(double));

    // Copy the data from the host memory to the device memory using
    // hipMemcpy
    hipMemcpy(device_sampleSeq, host_sampleSeq, totalSampleSize * sizeof(char),
               hipMemcpyHostToDevice);
    hipMemcpy(device_signatureSeq, host_signatureSeq,
               totalSignatureSize * sizeof(char), hipMemcpyHostToDevice);
    hipMemcpy(device_sampleQual, host_sampleQual,
               totalSampleSize * sizeof(char), hipMemcpyHostToDevice);
    hipMemcpy(device_samplesSize, host_sampleSize, sampleNum * sizeof(int),
               hipMemcpyHostToDevice);
    hipMemcpy(device_signaturesSize, host_signatureSize,
               signatureNum * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(device_sampleIdx, host_sampleIdx, sampleNum * sizeof(int),
               hipMemcpyHostToDevice);
    hipMemcpy(device_signatureIdx, host_signatureIdx,
               signatureNum * sizeof(int), hipMemcpyHostToDevice);

    // 3. Then run the kernel to find the match
    // Initialising the block size and grid size
    int blockSize = 256;
    int gridSize = ((sampleNum * signatureNum) + blockSize - 1) / blockSize;
    matchKernel<<<gridSize, blockSize>>>(
        device_sampleSeq, device_signatureSeq, device_sampleQual,
        device_samplesSize, device_signaturesSize, device_matchValue, sampleNum,
        signatureNum, device_sampleIdx, device_signatureIdx);

    // 4. Then copy the data from the device memory to the host memory
    hipMemcpy(host_matchValue, device_matchValue,
               sampleNum * signatureNum * sizeof(double),
               hipMemcpyDeviceToHost);
    // printing out all the match values
    // for (int i = 0; i < sampleNum; i++) {
    //     for (int j = 0; j < signatureNum; j++) {
    //         printf("Match value: %f\n", host_matchValue[i * signatureNum
    //         + j]);
    // 5. Then copy the data from the host memory to the vector of
    // MatchResult
    for (int i = 0; i < sampleNum; i++) {
        for (int j = 0; j < signatureNum; j++) {
            if (host_matchValue[i * signatureNum + j] != 0) {
                std::string currentSampleName = samples[i].name;
                std::string currentSignatureName = signatures[j].name;
                MatchResult matchResult = {
                    currentSampleName, currentSignatureName,
                    host_matchValue[i * signatureNum + j]};
                matches.push_back(matchResult);
            }
        }
    }
    // 6. Finally, free the memory
    free(host_sampleSeq);
    free(host_signatureSeq);
    free(host_sampleQual);
    free(host_sampleSize);
    free(host_signatureSize);
    free(host_sampleIdx);
    free(host_signatureIdx);
    free(host_matchValue);
    hipFree(device_sampleSeq);
    hipFree(device_signatureSeq);
    hipFree(device_sampleQual);
    hipFree(device_samplesSize);
    hipFree(device_signaturesSize);
    hipFree(device_sampleIdx);
    hipFree(device_signatureIdx);
    hipFree(device_matchValue);
}
